#include "hip/hip_runtime.h"
/**
 * @file CosmicUtils.cuh
 * @author Michal Solanik
 * @brief Implementation of common functions for simulations.
 * @version 0.1
 * @date 2021-07-15
 * 
 * @copyright Copyright (c) 2021
 * 
 */

#include "CosmicUtils.cuh"
#include "CosmicConstants.cuh"

__device__ int outputCounter = 0;

__device__ float getTkinInjection(unsigned long long state)
{
	unsigned long long ownState = state;
	int modulo;
	if (ownState > (injectionMax * quantityPerEnergy))
	{
		ownState -= (__double2ll_rd(ownState / (injectionMax * quantityPerEnergy)) * (injectionMax * quantityPerEnergy));
	}
	if (ownState >= quantityPerEnergy)
	{
		modulo = __double2int_rd(ownState / quantityPerEnergy);
	}
	else
	{
		modulo = 0;
	}
	return ((modulo) + ((ownState - (modulo * quantityPerEnergy) + 1) / quantityPerEnergy));
}

__device__ float getSolarPropInjection(unsigned long long state)
{
	int modulo = state % 30;
	return 0.01f * powf((1.0f + 0.5f), modulo); 
}

__global__ void nullCount()
{
	outputCounter = 0;
}

__global__ void curandInitialization(hiprandState_t *state)
{
	int execID = blockIdx.x * blockDim.x + threadIdx.x;
	hiprand_init(clock(), execID, 0, &state[execID]);
}