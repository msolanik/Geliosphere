#include "hip/hip_runtime.h"
/**
 * @file CosmicUtils.cuh
 * @author Michal Solanik
 * @brief Implementation of common functions for simulations.
 * @version 0.1
 * @date 2021-07-15
 * 
 * @copyright Copyright (c) 2021
 * 
 */

#include "CosmicUtils.cuh"
#include "CosmicConstants.cuh"

extern "C" void initRNG(hiprandState_t *state, int blockSize, int threadSize);

__device__ int outputCounter = 0;

__device__ float getTkinInjection(unsigned long long state)
{
	unsigned long long ownState = state;
	int modulo;
	if (ownState > (injectionMax * quantityPerEnergy))
	{
		ownState -= (__double2ll_rd(ownState / (injectionMax * quantityPerEnergy)) * (injectionMax * quantityPerEnergy));
	}
	if (ownState >= quantityPerEnergy)
	{
		modulo = __double2int_rd(ownState / quantityPerEnergy);
	}
	else
	{
		modulo = 0;
	}
	return ((modulo) + ((ownState - (modulo * quantityPerEnergy) + 1) / quantityPerEnergy));
}

__global__ void nullCount()
{
	outputCounter = 0;
}

__global__ void curandInitialization(hiprandState_t *state)
{
	int execID = blockIdx.x * blockDim.x + threadIdx.x;
	hiprand_init(clock(), execID, 0, &state[execID]);
}

void initRNG(hiprandState_t *state, int blockSize, int threadSize)
{
	curandInitialization<<<blockSize, threadSize>>>(state);
	gpuErrchk(hipDeviceSynchronize());
}