#include "hip/hip_runtime.h"
/**
 * @file OneDimensionBpSimulation.cu
 * @author Michal Solanik
 * @brief Implementation of 1D B-p method.
 * @version 0.1
 * @date 2021-07-14
 * 
 * @copyright Copyright (c) 2021
 * 
 */

#include <stdio.h>
#include <math.h>
#include <string>

#include "spdlog/spdlog.h"

#include "ParamsCarrier.hpp"
#include "FileUtils.hpp"
#include "OneDimensionBpSimulation.cuh"
#include "CosmicConstants.cuh"
#include "CosmicUtils.cuh"
#include "CudaErrorCheck.cuh"

/**
 * @brief Calculate pre-simulations parameters.
 * 
 * @param Tkininj Injecting kinetic energy.
 * @param pinj Injecting particle momentum.
 * @param padding Support value used to calculate state for getting
 * kinetic energy.
 */
__global__ void wCalcBP(float *Tkininj, float *pinj, int padding)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	float Tkin = getTkinInjection(BLOCK_SIZE_BP * THREAD_SIZE_BP * padding + id);
	float Rig = sqrtf(Tkin * (Tkin + (2 * T0)));
	float p = Rig * 1e9 * q / c;
	pinj[id] = p;
	Tkininj[id] = Tkin;
}

/**
 * @brief Run simulations for 1D B-p method. 
 * More information about approach choosed for 1D B-p model can be found here:
 * https://agupubs.onlinelibrary.wiley.com/doi/pdfdirect/10.1002/2015JA022237
 * 
 * @param pinj Injecting particle momentum.
 * @param history Data structure containing output records.
 * @param padding Support value used to calculate state for getting
 * kinetic energy.
 * @param state Array of random number generator data structures.
 */
__global__ void trajectorySimulationBP(float *pinj, trajectoryHistoryBP *history, int padding, hiprandState *state)
{
	extern __shared__ int sharedMemory[];
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	int idx = threadIdx.x;
	float r = 1.0f;
	float p = pinj[id];
	float beta, Rig, dr, pp;
	float Tkin = getTkinInjection(BLOCK_SIZE_BP * THREAD_SIZE_BP * padding + id);
	float2 *generated = (float2 *)sharedMemory;
	hiprandState *cuState = (hiprandState *)(&generated[THREAD_SIZE_BP]);
	cuState[idx] = state[blockIdx.x * blockDim.x + threadIdx.x];
	int count;
	bool generate = true;
	for (; r < 100.0002f;)
	{
		beta = sqrtf(Tkin * (Tkin + T0 + T0)) / (Tkin + T0);
		Rig = (p * c / q) / 1e9f;
		pp = p;
		p -= (2.0f * V * pp * dt / (3.0f * r));
		if (generate)
		{
			generated[idx] = curand_box_muller(&cuState[idx]);
			dr = (V + (2.0f * K0 * beta * Rig / r)) * dt + (generated[idx].x * sqrtf(2.0f * K0 * beta * Rig * dt));
			r += dr;
			generate = false;
		}
		else
		{
			dr = (V + (2.0f * K0 * beta * Rig / r)) * dt + (generated[idx].y * sqrtf(2.0f * K0 * beta * Rig * dt));
			r += dr;
			generate = true;
		}
		Rig = p * c / q;
		Tkin = (sqrtf((T0 * T0 * q * q * 1e9f * 1e9f) + (q * q * Rig * Rig)) - (T0 * q * 1e9f)) / (q * 1e9f);
		Rig = Rig / 1e9f;
		beta = sqrtf(Tkin * (Tkin + T0 + T0)) / (Tkin + T0);
		if (beta > 0.01f && Tkin < 200.0f)
		{
			if ((r > 100.0f) && ((r - dr) < 100.0f))
			{
				count = atomicAdd(&outputCounter, 1);
				double newW = (m0_double * m0_double * c_double * c_double * c_double * c_double) + (p * p * c_double * c_double);
				newW = (pow(newW, -1.85) / p) / 1e45;
				history[count].setValues(Tkin, r, newW, id);
				break;
			}
		}
		else if (beta < 0.01f)
		{
			break;
		}
		if (r < 0.3f)
		{
			r -= dr;
			p = pp;
		}
	}
	state[id] = cuState[idx];
}

/**
 * @brief Run 1D B-p method with given parameters defines 
 * in input simulation data structure.
 * 
 */
void runBPMethod(simulationInputBP *simulation)
{
	int counter;
	ParamsCarrier *singleTone;
	singleTone = simulation->singleTone;
    spdlog::info("Starting initialization of 1D B-p simulation.");
	
	std::string destination = singleTone->getString("destination", "");
	if (destination.empty())
	{
		destination = getDirectoryName(singleTone);
        spdlog::info("Destination is not specified - using generated name for destination: " + destination);
	}
	if (!createDirectory("BP", destination))
	{
        spdlog::error("Directory for 1D B-p simulations cannot be created.");
		return;
	}

	FILE *file = fopen("log.dat", "w");
	curandInitialization<<<simulation->blockSize, simulation->threadSize>>>(simulation->state);
	gpuErrchk(hipDeviceSynchronize());
	int iterations = ceil((float)singleTone->getInt("millions", 1) * 1000000 / ((float)simulation->blockSize * (float)simulation->threadSize));
	if (simulation->threadSize == 1024)
	{
		hipFuncSetAttribute(reinterpret_cast<const void*>(trajectorySimulationBP), hipFuncAttributeMaxDynamicSharedMemorySize, 65536);
	}
	for (int k = 0; k < iterations; ++k)
	{
        spdlog::info("Processed: {:03.2f}%", (float)k / ((float)iterations / 100.0));
		nullCount<<<1, 1>>>();
		gpuErrchk(hipDeviceSynchronize());
		wCalcBP<<<simulation->blockSize, simulation->threadSize>>>(simulation->Tkininj, simulation->pinj, k);
		gpuErrchk(hipDeviceSynchronize());
		trajectorySimulationBP<<<simulation->blockSize, simulation->threadSize, simulation->threadSize * sizeof(hiprandState_t) + simulation->threadSize * sizeof(float2)>>>(simulation->pinj, simulation->history, k, simulation->state);
		gpuErrchk(hipDeviceSynchronize());
		hipMemcpyFromSymbol(&counter, HIP_SYMBOL(outputCounter), sizeof(int), 0, hipMemcpyDeviceToHost);
        spdlog::info("In this iteration {} particles were detected.", counter);
		if (counter != 0)
		{
			gpuErrchk(hipMemcpy(simulation->local_history, simulation->history, counter * sizeof(trajectoryHistoryBP), hipMemcpyDeviceToHost));
			for (int j = 0; j < counter; ++j)
			{
				fprintf(file, "%g %g %g %g\n", simulation->local_history[j].Tkin, simulation->Tkininj[simulation->local_history[j].id],
						simulation->local_history[j].r, simulation->local_history[j].w);
			}
		}
	}
	fclose(file);
    spdlog::info("Simulation ended.");
}